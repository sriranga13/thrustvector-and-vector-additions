#include "hip/hip_runtime.h"
/* ACADEMIC INTEGRITY PLEDGE                                              */
/*                                                                        */
/* - I have not used source code obtained from another student nor        */
/*   any other unauthorized source, either modified or unmodified.        */
/*                                                                        */
/* - All source code and documentation used in my program is either       */
/*   my original work or was derived by me from the source code           */
/*   published in the textbook for this course or presented in            */
/*   class.                                                               */
/*                                                                        */
/* - I have not discussed coding details about this project with          */
/*   anyone other than my instructor. I understand that I may discuss     */
/*   the concepts of this program with other students and that another    */
/*   student may help me debug my program so long as neither of us        */
/*   writes anything during the discussion or modifies any computer       */
/*   file during the discussion.                                          */
/*                                                                        */
/* - I have violated neither the spirit nor letter of these restrictions. */
/*                                                                        */
/*                                                                        */
/*                                                                        */
/* Signed:Sriranga    Date:3/4/21      */
/*                                                                        */
/*                                                                        */
/* 3460:677 CUDA Vector Add lab, Version 1.01, Fall 2016.                 */

#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
#include <string.h>
#include <hip/hip_runtime.h>


__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
        int bid=blockIdx.x+gridDim.x*+blockIdx.y;
	int tid= blockDim.x*bid+threadIdx.x;
	if(tid<len)
	{
             out[tid]=in1[tid]+in2[tid];
	}

}

int main(int argc, char **argv) {
  int inputLength1, inputLength2,outputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;
  float *expectedOutput;

  FILE *infile1, *infile2, *outfile;
  unsigned int generic, gpu, copy, compute, blog = 1;

  // Import host input data
  CUT_SAFE_CALL(cutCreateTimer(&generic));
  cutStartTimer(generic);
  if ((infile1 = fopen("input0.raw", "r")) == NULL)
  { printf("Cannot open input0.raw.\n"); exit(EXIT_FAILURE); }
  if ((infile2 = fopen("input1.raw", "r")) == NULL)
  { printf("Cannot open input1.raw.\n"); exit(EXIT_FAILURE); }
  fscanf(infile1, "%i", &inputLength1);
  hostInput1 = (float*) malloc(sizeof(float) * inputLength1);
  for (int i = 0; i < inputLength1; i++)
    fscanf(infile1, "%f", &hostInput1[i]);
  fscanf(infile2, "%i", &inputLength2);
  hostInput2 = (float*) malloc(sizeof(float) * inputLength2);
  for (int i = 0; i < inputLength2; i++)
    fscanf(infile2, "%f", &hostInput2[i]);
  fclose(infile1);
  fclose(infile2);
  hostOutput = (float *)malloc(sizeof(float) * inputLength1);
  cutStopTimer(generic);
  printf("Importing data and creating memory on host: %f ms\n", cutGetTimerValue(generic));

  if (blog) printf("*** The input length is %i\n", inputLength1);

  CUT_SAFE_CALL(cutCreateTimer(&gpu));
  cutStartTimer(gpu);
  //@@ Allocate GPU memory here

  int size=inputLength1*sizeof(float);
  outputLength=inputLength1;
  hipMalloc(&deviceInput1,size);
  hipMalloc(&deviceInput2,size);
  hipMalloc(&deviceOutput,size);
  cutStopTimer(gpu);
  printf("Allocating GPU memory: %f ms\n", cutGetTimerValue(gpu));

  cutDeleteTimer(gpu);
  CUT_SAFE_CALL(cutCreateTimer(&gpu));
  cutStartTimer(gpu);

  //@@ Copy memory to the GPU here
   hipMemcpy(deviceInput1,hostInput1,size,hipMemcpyHostToDevice);
   hipMemcpy(deviceInput2,hostInput2,size,hipMemcpyHostToDevice);
  
  cutStopTimer(gpu);
  printf("Copying input memory to the GPU: %f ms\n", cutGetTimerValue(gpu));

  //@@ Initialize the grid and block dimensions here
 
  dim3 blockDim(256);
  dim3 gridDim(1);

  if (blog) printf("*** Block dimension is %i\n", blockDim.x);
  if (blog) printf("*** Grid dimension is %i\n", gridDim.x);

  CUT_SAFE_CALL(cutCreateTimer(&compute));
  cutStartTimer(compute);
  
  //@@ Launch the GPU Kernel here
  vecAdd<<<gridDim,blockDim>>>(deviceInput1,deviceInput2,deviceOutput,size);

  hipDeviceSynchronize();
  
  cutStopTimer(compute);
  printf("Performing CUDA computation: %f ms\n", cutGetTimerValue(compute));

  CUT_SAFE_CALL(cutCreateTimer(&copy));
  cutStartTimer(copy);

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput,deviceOutput,size, hipMemcpyDeviceToHost);
  
  cutStopTimer(copy);
  printf("Copying output memory to the CPU: %f ms\n", cutGetTimerValue(copy));

  cutDeleteTimer(gpu);
  CUT_SAFE_CALL(cutCreateTimer(&gpu));
  cutStartTimer(gpu);
  
  //@@ Free the GPU memory here
   hipFree(deviceInput1);
   hipFree(deviceInput2);
   hipFree(deviceOutput);
  
  cutStopTimer(gpu);
  printf("Freeing GPU Memory: %f ms\n", cutGetTimerValue(gpu));

  if ((outfile = fopen("output.raw", "r")) == NULL)
  { printf("Cannot open output.raw.\n"); exit(EXIT_FAILURE); }
  fscanf(outfile, "%i", &outputLength);
  expectedOutput = (float*) malloc(sizeof(float) * outputLength);
  for (int i = 0; i < outputLength; i++)
    fscanf(outfile, "%f", &expectedOutput[i]);
  fclose(outfile);
  int test = 1;
  for (int i = 0; i < outputLength; i++)
    test = test && (abs(expectedOutput[i] - hostOutput[i]) < 0.005);
  if (test) printf("Results correct.\n");
  else printf("Results incorrect.\n");

  cutDeleteTimer(generic);
  cutDeleteTimer(gpu);
  cutDeleteTimer(copy);
  cutDeleteTimer(compute);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(expectedOutput);
  
  return 0;
}
