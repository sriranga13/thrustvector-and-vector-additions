#include "hip/hip_runtime.h"
/* ACADEMIC INTEGRITY PLEDGE                                              */
/*                                                                        */
/* - I have not used source code obtained from another student nor        */
/*   any other unauthorized source, either modified or unmodified.        */
/*                                                                        */
/* - All source code and documentation used in my program is either       */
/*   my original work or was derived by me from the source code           */
/*   published in the textbook for this course or presented in            */
/*   class.                                                               */
/*                                                                        */
/* - I have not discussed coding details about this project with          */
/*   anyone other than my instructor. I understand that I may discuss     */
/*   the concepts of this program with other students and that another    */
/*   student may help me debug my program so long as neither of us        */
/*   writes anything during the discussion or modifies any computer       */
/*   file during the discussion.                                          */
/*                                                                        */
/* - I have violated neither the spirit nor letter of these restrictions. */
/*                                                                        */
/*                                                                        */
/*                                                                        */
/* Signed:Sriranga Date:3/5/2021       */
/*                                                                        */
/*                                                                        */
/* 3460:677 CUDA Thrust Vector Add lab, Version 1.01, Fall 2016.          */

#include <stdio.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cutil.h>

int main(int argc, char *argv[]) {
  float *hostInput1;
  float *hostInput2;
  float *expectedOutput;
  int inputLength1, inputLength2, outputLength;

  FILE *infile1, *infile2, *outfile;
  unsigned int generic, gpu1, gpu2, copy, compute;

  // Import host input data
  CUT_SAFE_CALL(cutCreateTimer(&generic));
  cutStartTimer(generic);
  if ((infile1 = fopen("input0.raw", "r")) == NULL)
  { printf("Cannot open input0.raw.\n"); exit(EXIT_FAILURE); }
  if ((infile2 = fopen("input1.raw", "r")) == NULL)
  { printf("Cannot open input1.raw.\n"); exit(EXIT_FAILURE); }
  fscanf(infile1, "%i", &inputLength1);
  hostInput1 = (float*) malloc(sizeof(float) * inputLength1);
  for (int i = 0; i < inputLength1; i++)
    fscanf(infile1, "%f", &hostInput1[i]);
  fscanf(infile2, "%i", &inputLength2);
  hostInput2 = (float*) malloc(sizeof(float) * inputLength2);
  for (int i = 0; i < inputLength2; i++)
    fscanf(infile2, "%f", &hostInput2[i]);
  fclose(infile1);
  fclose(infile2);
  cutStopTimer(generic);
  printf("Importing data to host: %f ms\n", cutGetTimerValue(generic));

  // Allocate host output vector
  //@@ Insert code here

  float *hostOutput;
  outputLength = inputLength1;
  hostOutput = (float *)malloc(sizeof(float)*outputLength);
  CUT_SAFE_CALL(cutCreateTimer(&gpu1));
  cutStartTimer(gpu1);

  CUT_SAFE_CALL(cutCreateTimer(&gpu2));
  cutStartTimer(gpu2);

  // Declare and allocate thrust device input and output vectors
  //@@ Insert code here

  thrust::device_vector<float>deviceInput1(inputLength1);
  thrust::device_vector<float>deviceInput2(inputLength2);
  thrust::device_vector<float>deviceOutput(outputLength);
  cutStopTimer(gpu2);
  printf("Doing GPU memory allocation: %f ms\n", cutGetTimerValue(gpu2));

  CUT_SAFE_CALL(cutCreateTimer(&copy));
  cutStartTimer(copy);

  // Copy to device
  //@@ Insert code here

  thrust::copy(hostInput1, hostInput1 + inputLength1 , deviceInput1.begin());
  thrust::copy(hostInput2, hostInput2 + inputLength2 , deviceInput2.begin());
  cutStopTimer(copy);
  printf("Copying data to the GPU: %f ms\n", cutGetTimerValue(copy));

  CUT_SAFE_CALL(cutCreateTimer(&compute));
  cutStartTimer(compute);

  // Execute vector addition
  //@@ Insert Code here

  thrust::transform(deviceInput1.begin(),deviceInput1.end(),deviceInput2.begin(),deviceOutput.begin(),thrust::plus<float>());
  cutStopTimer(compute);
  printf("Doing the computation on the GPU: %f ms\n", cutGetTimerValue(compute));
  /////////////////////////////////////////////////////////

  cutDeleteTimer(copy);
  CUT_SAFE_CALL(cutCreateTimer(&copy));
  cutStartTimer(copy);

  // Copy data back to host
  //@@ Insert code here
  
  thrust::copy(deviceOutput.begin(),deviceOutput.end(),hostOutput);
  cutStopTimer(copy);
  printf("Copying data from the GPU: %f ms\n", cutGetTimerValue(copy));

  cutStopTimer(gpu1);
  printf("Doing GPU computation (memory + compute): %f ms\n", cutGetTimerValue(gpu1));

  if ((outfile = fopen("output.raw", "r")) == NULL)
  { printf("Cannot open output.raw.\n"); exit(EXIT_FAILURE); }
  fscanf(outfile, "%i", &outputLength);
  expectedOutput = (float*) malloc(sizeof(float) * outputLength);
  for (int i = 0; i < outputLength; i++)
    fscanf(outfile, "%f", &expectedOutput[i]);
  fclose(outfile);
  int test = 1;
  for (int i = 0; i < outputLength; i++)
    test = test && (abs(expectedOutput[i] - hostOutput[i]) < 0.005);
  if (test) printf("Results correct.\n");
  else printf("Results incorrect.\n");

  cutDeleteTimer(generic);
  cutDeleteTimer(gpu1);
  cutDeleteTimer(gpu2);
  cutDeleteTimer(copy);
  cutDeleteTimer(compute);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(expectedOutput);
  return 0;
}
